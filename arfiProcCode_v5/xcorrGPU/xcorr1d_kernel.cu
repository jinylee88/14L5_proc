#include "hip/hip_runtime.h"
__global__ void xcorr1d_kernel(float *disp, float *cc, int nsamps, int nsteps, int nbeams, int srchsz, int kernsz, int blk) {

	int samp = threadIdx.x +    blk    *blockDim.x; 
	int step = threadIdx.y + blockIdx.y*blockDim.y;
	int beam = threadIdx.z + blockIdx.z*blockDim.z;

	int halfk = (kernsz-1)/2;
	int range = (srchsz - kernsz)/2;

	if (samp < nsamps && step < nsteps-1 && beam < nbeams) {

		disp += nsamps*(step + (nsteps-1)*beam);
		
		float x, y, sum_X, sum_Y, sumXX, sumYY, sumXY, tmp, maxrho, d;
		d = 0.0f; maxrho = -1.0f;

		// Build off of previous displacement, if available
		float prev_d = 0.0f;
		if (blk > 0) prev_d = disp[blk*blockDim.x-1];

		// Load post-track data
		sum_Y = sumYY = 0.0f;
		for (int k = -halfk; k <= halfk; k++) {
		
			// Fetch data from texture
			y = tex2D(rf_tex, samp+k + 0.5f, step+1 + beam*nsteps + 0.5f);
			
			// Accumulate sums
			sum_Y += y;
			sumYY += y*y;
		}

		// Loop through search region of reference track and compute CC's
		for (float s = samp-range+prev_d; s <= samp+range+prev_d; s+=0.05f) {
			sum_X = sumXX = sumXY = 0.0f;

			// Loop over kernel
			for (int k = -halfk; k <= halfk; k++) {

				// Fetch data from texture
				x = tex2D(rf_tex, s+k    + 0.5f, step+0 + beam*nsteps + 0.5f);
				y = tex2D(rf_tex, samp+k + 0.5f, step+1 + beam*nsteps + 0.5f);

				// Accumulate sums
				sum_X += x;
				sumXX += x*x;
				sumXY += x*y;

			}
			// Compute normalized cross correlation coefficient
  	        tmp = (kernsz*sumXY - sum_X*sum_Y) *
 			 rsqrt((kernsz*sumXX - sum_X*sum_X) *
 			       (kernsz*sumYY - sum_Y*sum_Y));
			
			if (!isinf(tmp) && tmp > maxrho) {
				maxrho = tmp;
				d      = s-samp;
			}
		}

		// Store outputs
		disp[samp] = d;
		cc  [samp] = maxrho;
	}
} 
/*
__global__ void xcorr1d_kernel(float *disp, float *cc, int nsamps, int nsteps, int srchsz, int kernsz) {

	int samp = threadIdx.x + blockIdx.x*blockDim.x; 
	int step = threadIdx.y + blockIdx.y*blockDim.y;

	int halfk = (kernsz-1)/2;
	int range = (srchsz - kernsz)/2;

	if (samp < nsamps && step < nsteps) {
		
		float x, y, sum_X, sum_Y, sumXX, sumYY, sumXY, tmp, maxrho, d;
		d = 0.0f; maxrho = -1.0f;
		
		for (float s = samp-range; s <= samp+range; s+=0.05f) {

			sum_X = sum_Y = sumXX = sumYY = sumXY = 0.0f;
			for (int k = -halfk; k <= halfk; k++) {
				
				// Fetch data from texture
				x = tex2D(rf_tex, s+k    + 0.5f, 0    + 0.5f);
				y = tex2D(rf_tex, samp+k + 0.5f, step + 0.5f);

				// Accumulate sums
				sum_X += x;
				sum_Y += y;
				sumXX += x*x;
				sumYY += y*y;
				sumXY += x*y;

			}
			// Compute normalized cross correlation coefficient
  	        tmp = (kernsz*sumXY - sum_X*sum_Y) *
 			 rsqrt((kernsz*sumXX - sum_X*sum_X) *
 			       (kernsz*sumYY - sum_Y*sum_Y));
			
			if (!isinf(tmp) && tmp > maxrho) {
				maxrho = tmp;
				d      = s-samp;
			}
		}

		disp[samp + step*nsamps] = d;
		cc  [samp + step*nsamps] = maxrho;

	}
} 
*/
