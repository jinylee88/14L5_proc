#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Code-specific includes
#include <mex.h>

// Declare texture reference for 2D float texture
texture<float, hipTextureType2D, hipReadModeElementType> rf_tex;

// Function prototypes
void cleanup();
void xcorr(int nlhs, mxArray *plhs[], float *rfdata, int nsamps, int nsteps, int nbeams, int srchsz, int kernsz);

// CUDA kernels
#include "xcorr1d_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Gateway function to MATLAB (main function)
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

	if (nrhs != 3)
		mexErrMsgTxt("Wrong number of inputs.\n");
	if (mxGetClassID(prhs[0]) != mxDOUBLE_CLASS && mxGetClassID(prhs[0]) != mxSINGLE_CLASS)
		mexErrMsgTxt("The input rfdata must be of class single or double.\n");
	float *rfdata;

//	if (mxGetNumberOfDimensions(prhs[0]) > 2)
//		mexErrMsgTxt("Only have functionality for 1D xcorr for now. Input must be in (fast time x slow time).\n");

	if (mxGetClassID(prhs[0]) == mxDOUBLE_CLASS) {
		int numel = mxGetNumberOfElements(prhs[0]);
		double *dat = mxGetPr(prhs[0]);
		rfdata = (float *)mxMalloc(sizeof(float)*numel);
		for (int i = 0; i < numel; i++)
			rfdata[i] = (float)dat[i];
	}
	else
		rfdata = (float *)mxGetData(prhs[0]);
	int srchsz = mxGetScalar(prhs[1]);
	int kernsz = mxGetScalar(prhs[2]);

	// Get dimensions of data
	mwSize ndims = mxGetNumberOfDimensions(prhs[0]);
	const mwSize *dims;
	dims = mxGetDimensions(prhs[0]);
	int nsamps = dims[0];
	int nsteps = dims[1];
	int nbeams = 1; for (int i = 2; i < ndims; i++) nbeams *= dims[i];

	// Run cross-correlation peak detector script
	xcorr(nlhs, plhs, rfdata, nsamps, nsteps, nbeams, srchsz, kernsz);
	
	if (mxGetClassID(prhs[0]) == mxDOUBLE_CLASS)
		mxFree(rfdata);
	
	hipDeviceReset();
}

void xcorr(int nlhs, mxArray *plhs[], float *rfdata, int nsamps, int nsteps, int nbeams, int srchsz, int kernsz) {

	hipChannelFormatDesc channelDescFLOAT = hipCreateChannelDesc<float>();
	rf_tex.addressMode[0] = hipAddressModeClamp;
	rf_tex.addressMode[1] = hipAddressModeClamp;
	rf_tex.filterMode     = hipFilterModeLinear;
	rf_tex.normalized     = false;

	float *rf_d, *disp_d, *ccs_d;
	size_t pitch;
	hipMallocPitch(&rf_d, &pitch, sizeof(float)*nsamps, nsteps*nbeams);
	hipMalloc((void **)&disp_d,sizeof(float)*nsamps*(nsteps-1)*nbeams);
	hipMalloc((void **)&ccs_d, sizeof(float)*nsamps*(nsteps-1)*nbeams);
	hipMemcpy2D(rf_d, pitch, rfdata, sizeof(float)*nsamps,
			sizeof(float)*nsamps, nsteps*nbeams, hipMemcpyHostToDevice);
	hipBindTexture2D(NULL, rf_tex, rf_d, channelDescFLOAT, nsamps, nsteps*nbeams, pitch);
	hipMemset(disp_d, 0, sizeof(float)*nsamps*(nsteps-1)*nbeams);

	dim3 dimB(16, 16, 1);
	dim3 dimG(1, ceil(nsteps/dimB.y)+1, ceil(nbeams/dimB.z)+1);	

	for (int blk = 0; blk < ceil(nsamps/dimB.x)+1; blk++) {
		xcorr1d_kernel<<<dimG, dimB, 0>>>(disp_d, ccs_d, nsamps, nsteps, nbeams, srchsz, kernsz, blk);
		hipDeviceSynchronize();
	}

	mwSize dims[3] = {nsamps, nsteps-1, nbeams};
	plhs[0] = mxCreateNumericArray(3, dims, mxSINGLE_CLASS, mxREAL);
	plhs[1] = mxCreateNumericArray(3, dims, mxSINGLE_CLASS, mxREAL);

	float *disp_h, *ccs_h;
	disp_h = (float *)mxGetData(plhs[0]);
	ccs_h  = (float *)mxGetData(plhs[1]);
	
	hipUnbindTexture(rf_tex);
	hipFree(rf_d);
	hipMemcpy(disp_h, disp_d,sizeof(float)*nsamps*(nsteps-1)*nbeams, hipMemcpyDeviceToHost);
	hipMemcpy(ccs_h,  ccs_d, sizeof(float)*nsamps*(nsteps-1)*nbeams, hipMemcpyDeviceToHost);

}



















































